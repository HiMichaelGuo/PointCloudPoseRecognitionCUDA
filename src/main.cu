
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void kernel() {
    printf("Just a test! I am point cloud %d!\n", blockIdx.x);
}
int main() {
    kernel<<<9, 1>>>();
    hipDeviceSynchronize(); ///wait for the kernel function to finish the execution, and then continue to execute the following code
    return 1;
}